#include "hip/hip_runtime.h"
#define SOLVER_EXPORTS

#include "Solver.h"
#include "CudaUnitility.h"
#include <>

static __device__ int3 combine_int3(int a, int b, int c)
{
	int3 res;
	res.x = a, res.y = b, res.z = c;
	return res;
}

static __device__ float3 combine_float3(float a, float b, float c)
{
	float3 res;
	res.x = a, res.y = b, res.z = c;
	return res;
}

template<typename T>
static __device__ T min(T a, T b)
{
	return a < b ? a : b;
}

template<typename T>
static __device__ T max(T a, T b)
{
	return a > b ? a : b;
}

static __device__ int3 minmax(int3 pos, int3 max_pos)
{
	pos.x = max(0, min(pos.x, max_pos.x - 1));
	pos.y = max(0, min(pos.y, max_pos.y - 1));
	pos.z = max(0, min(pos.z, max_pos.z - 1));
	return pos;
}

static __device__ float sample(float* field, int3 pos, int3 max_pos)
{
	pos = minmax(pos, max_pos);
	return field[pos.x + pos.y * max_pos.x + pos.z * max_pos.x * max_pos.y];
}

static __device__ float cg_sample(float* field, int3 pos, int3 max_pos)
{
	if (pos.x < 0 || pos.x >= max_pos.x || pos.y < 0 || pos.y >= max_pos.y || pos.z < 0 || pos.z >= max_pos.z)
		return 0;
	return field[pos.x + pos.y * max_pos.x + pos.z * max_pos.x * max_pos.y];
}

static __device__ float length(float3 f)
{
	return sqrt(pow(f.x, 2) + pow(f.y, 2) + pow(f.z, 2));
}

static __device__ float3 normalize(float3 f)
{
	float len = length(f) + 1e-5;
	return combine_float3(f.x / len, f.y / len, f.z / len);
}

static __device__ float3 cross(float3 a, float3 b)
{
	return combine_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

static __device__ float lerp(float a, float b, float s)
{
	return a * (1 - s) + b * s;
}

static __device__ float trilerp(float* field, float3 pos, int3 max_pos)
{
	float x = pos.x;
	float y = pos.y;
	float z = pos.z;

	int ix = int(x);
	int iy = int(y);
	int iz = int(z);

	float fx = x - floor(x);
	float fy = y - floor(y);
	float fz = z - floor(z);

	float a = sample(field, combine_int3(ix, iy, iz), max_pos);
	float b = sample(field, combine_int3(ix + 1, iy, iz), max_pos);
	float c = sample(field, combine_int3(ix, iy + 1, iz), max_pos);
	float d = sample(field, combine_int3(ix + 1, iy + 1, iz), max_pos);
	float e = sample(field, combine_int3(ix, iy, iz + 1), max_pos);
	float f = sample(field, combine_int3(ix + 1, iy, iz + 1), max_pos);
	float g = sample(field, combine_int3(ix, iy + 1, iz + 1), max_pos);
	float h = sample(field, combine_int3(ix + 1, iy + 1, iz + 1), max_pos);

	float lerp1 = lerp(lerp(a, b, fx), lerp(c, d, fx), fy);
	float lerp2 = lerp(lerp(e, f, fx), lerp(g, h, fx), fy);

	return lerp(lerp1, lerp2, fz);
}

static __device__ float3 operator*(float a, float3 b)
{
	b.x *= a;
	b.y *= a;
	b.z *= a;
	return b;
}

static __device__ float3 operator*(float3 a, float3 b)
{
	b.x *= a.x;
	b.y *= a.y;
	b.z *= a.z;
	return b;
}

static __device__ float3 operator-(float3 a, float3 b)
{
	a.x -= b.x;
	a.y -= b.y;
	a.z -= b.z;
	return a;
}

static __device__ float3 RK2(float* ux, float* uy, float* uz, float3 pos, float dt, int3 max_pos)
{
	float3 u;
	u.x = trilerp(ux, pos, max_pos);
	u.y = trilerp(uy, pos, max_pos);
	u.z = trilerp(uz, pos, max_pos);
	float3 mid = pos - 0.5f * dt * u;
	u.x = trilerp(ux, mid, max_pos);
	u.y = trilerp(uy, mid, max_pos);
	u.z = trilerp(uz, mid, max_pos);
	// here may exist out of range problem
	return pos - dt * u;
}

// note that A and B here are one dimension vectors
static __device__ float aTb(float* a, float* b, int3 max_pos)
{
	float res = 0;
	for (int i = 0; i < max_pos.x * max_pos.y * max_pos.z; ++i)
		res += a[i] * b[i];
	return res;
}

static __device__ void device_swap(float** a, float** b)
{
	float* temp = *a;
	*a = *b;
	*b = temp;
}

void swap(float** a, float** b)
{
	float* temp = *a;
	*a = *b;
	*b = temp;
}

static __global__ void SourceKernel(float* rho, float* ux, float* uy, float* uz, float rho0, float3 u0)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	if (i > blockDim.x / 2 - 2 && i < blockDim.x / 2 + 2 && j > gridDim.x / 2 - 2 && j < gridDim.x / 2 + 2 && k > 1 && k < 3)
	{
		rho[ind] = rho0;
		ux[ind] = u0.x;
		uy[ind] = u0.y;
		uz[ind] = u0.z;
	}
}

static __global__ void TestKernel(float* rho, float* rho_new)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	if (ind > 0)
		rho_new[ind] = rho[ind - 1];
}

static __global__ void SemiLagKernel(float* field, float* new_field, float* ux, float* uy, float* uz, float dt, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float3 pos;
	pos.x = float(i);
	pos.y = float(j);
	pos.z = float(k);

	float3 coord = RK2(ux, uy, uz, pos, dt, max_pos);
	new_field[ind] = trilerp(field, coord, max_pos);
}

static __global__ void DivergenceKernel(float* field, float* ux, float* uy, float* uz, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float ul = sample(ux, combine_int3(i - 1, j, k), max_pos);
	float ur = sample(ux, combine_int3(i + 1, j, k), max_pos);
	float ubh = sample(uy, combine_int3(i, j - 1, k), max_pos);
	float uf = sample(uy, combine_int3(i, j + 1, k), max_pos);
	float ubo = sample(uz, combine_int3(i, j, k - 1), max_pos);
	float ut = sample(uz, combine_int3(i, j, k + 1), max_pos);

#if 0
	// box boundary
	float ucx = sample(ux, combine_int3(i, j, k), max_pos);
	float ucy = sample(uy, combine_int3(i, j, k), max_pos);
	float ucz = sample(uz, combine_int3(i, j, k), max_pos);
	if (i == 0)
		ul = -ucx;
	if (i == max_pos.x - 1)
		ur = -ucx;
	if (j == 0)
		ubh = -ucy;
	if (j == max_pos.y - 1)
		uf = -ucy;
	if (k == 0)
		ubo = -ucz;
	if (k == max_pos.z - 1)
		ut = -ucz;
#endif

	float div = (ur + uf + ut - ul - ubh - ubo) * 0.5;

	field[ind] = div;
}

static __global__ void JacobiKernel(float* field, float* new_field, float* div_field, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float pl = sample(field, combine_int3(i - 1, j, k), max_pos);
	float pr = sample(field, combine_int3(i + 1, j, k), max_pos);
	float pbh = sample(field, combine_int3(i, j - 1, k), max_pos);
	float pf = sample(field, combine_int3(i, j + 1, k), max_pos);
	float pbo = sample(field, combine_int3(i, j, k - 1), max_pos);
	float pt = sample(field, combine_int3(i, j, k + 1), max_pos);
	float div = sample(div_field, combine_int3(i, j, k), max_pos);

	new_field[ind] = (pl + pr + pbh + pf + pbo + pt - div) / 6.f;
}

static __global__ void ApplyGradient(float* f_ux, float* f_uy, float* f_uz, float* pressure_field, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float pl = sample(pressure_field, combine_int3(i - 1, j, k), max_pos);
	float pr = sample(pressure_field, combine_int3(i + 1, j, k), max_pos);
	float pbh = sample(pressure_field, combine_int3(i, j - 1, k), max_pos);
	float pf = sample(pressure_field, combine_int3(i, j + 1, k), max_pos);
	float pbo = sample(pressure_field, combine_int3(i, j, k - 1), max_pos);
	float pt = sample(pressure_field, combine_int3(i, j, k + 1), max_pos);

	f_ux[ind] -= 0.5 * (pr - pl);
	f_uy[ind] -= 0.5 * (pf - pbh);
	f_uz[ind] -= 0.5 * (pt - pbo);
}

static __global__ void VorticityKernel(float* f_vortx, float* f_vorty, float* f_vortz, float* ux, float* uy, float* uz, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float ul = sample(ux, combine_int3(i - 1, j, k), max_pos);
	float ur = sample(ux, combine_int3(i + 1, j, k), max_pos);
	float ubh = sample(uy, combine_int3(i, j - 1, k), max_pos);
	float uf = sample(uy, combine_int3(i, j + 1, k), max_pos);
	float ubo = sample(uz, combine_int3(i, j, k - 1), max_pos);
	float ut = sample(uz, combine_int3(i, j, k + 1), max_pos);

	f_vortx[ind] = (uf - ubh - ut + ubo) * 0.5;
	f_vorty[ind] = (ut - ubo - ur + ul) * 0.5;
	f_vortz[ind] = (ur - ul - uf + ubh) * 0.5;
}

static __global__ void ForceKernel(float* f_ux, float* f_uy, float* f_uz, float* f_vortx, float* f_vorty, float* f_vortz, float dt, float curl_strength, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float3 vl = combine_float3(sample(f_vortx, combine_int3(i - 1, j, k), max_pos),
		sample(f_vorty, combine_int3(i - 1, j, k), max_pos),
		sample(f_vortz, combine_int3(i - 1, j, k), max_pos));
	float3 vr = combine_float3(sample(f_vortx, combine_int3(i + 1, j, k), max_pos),
		sample(f_vorty, combine_int3(i + 1, j, k), max_pos),
		sample(f_vortz, combine_int3(i + 1, j, k), max_pos));
	float3 vbh = combine_float3(sample(f_vortx, combine_int3(i, j - 1, k), max_pos),
		sample(f_vorty, combine_int3(i, j - 1, k), max_pos),
		sample(f_vortz, combine_int3(i, j - 1, k), max_pos));
	float3 vf = combine_float3(sample(f_vortx, combine_int3(i, j + 1, k), max_pos),
		sample(f_vorty, combine_int3(i, j + 1, k), max_pos),
		sample(f_vortz, combine_int3(i, j + 1, k), max_pos));
	float3 vbo = combine_float3(sample(f_vortx, combine_int3(i, j, k - 1), max_pos),
		sample(f_vorty, combine_int3(i, j, k - 1), max_pos),
		sample(f_vortz, combine_int3(i, j, k - 1), max_pos));
	float3 vt = combine_float3(sample(f_vortx, combine_int3(i, j, k + 1), max_pos),
		sample(f_vorty, combine_int3(i, j, k + 1), max_pos),
		sample(f_vortz, combine_int3(i, j, k + 1), max_pos));
	float3 vc = combine_float3(sample(f_vortx, combine_int3(i, j, k), max_pos),
		sample(f_vorty, combine_int3(i, j, k), max_pos),
		sample(f_vortz, combine_int3(i, j, k), max_pos));

	// �� = ��|��|, N = ��/|��|
	float3 force = normalize(combine_float3(abs(length(vr)) - abs(length(vl)), abs(length(vf)) - abs(length(vbh)), abs(length(vt)) - abs(length(vbo))));
	// f_conf(vort) = ��h(N����)
	float3 fvort = curl_strength * cross(force, vc);

	f_ux[ind] += fvort.x * dt;
	f_uy[ind] += fvort.y * dt;
	f_uz[ind] += fvort.z * dt;
}

// -Ax = -b, r0 = -b = -����u, p0 = r0
static __global__ void InitConjugate(float* residual, float* p, float* f_div, float* f_pressure)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	residual[ind] = f_div[ind];
	p[ind] = f_div[ind];
	f_pressure[ind] = 0;
}

// p here is conjugate gradient, not pressure
static __global__ void ComputeAp(float* Ap, float* p, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float pl = cg_sample(p, combine_int3(i - 1, j, k), max_pos);
	float pr = cg_sample(p, combine_int3(i + 1, j, k), max_pos);
	float pbh = cg_sample(p, combine_int3(i, j - 1, k), max_pos);
	float pf = cg_sample(p, combine_int3(i, j + 1, k), max_pos);
	float pbo = cg_sample(p, combine_int3(i, j, k - 1), max_pos);
	float pt = cg_sample(p, combine_int3(i, j, k + 1), max_pos);
	float pc = cg_sample(p, combine_int3(i, j, k), max_pos);

	Ap[ind] = -6.f * pc + pl + pr + pbh + pf + pbo + pt;
}

static __global__ void UpdateResidual(float* residual, float* new_residual, float* p, float* Ap, float* f_pressure, float alpha)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	f_pressure[ind] += alpha * p[ind];
	new_residual[ind] = residual[ind] - alpha * Ap[ind];
}

static __global__ void UpdateP(float* p, float* new_residual, float beta)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	p[ind] = new_residual[ind] + beta * p[ind];
}

static __global__ void Conjugate(float* residual, float* new_residual, float* p, float* Ap, float* f_pressure, float* f_div, int3 max_pos)
{
	int nx = max_pos.x, ny = max_pos.y, nz = max_pos.z;
	InitConjugate << <dim3(ny, nz), nx >> > (residual, p, f_div, f_pressure);

	float init_rTr = aTb(residual, residual, max_pos);

	for (int i = 0; i < 40; ++i)
	{
		// ��(k) = r(k)Tr(k) / p(k)TAp(k)
		float alpha = aTb(residual, residual, max_pos);
		ComputeAp << <dim3(ny, nz), nx >> > (Ap, p, max_pos);
		alpha /= aTb(p, Ap, max_pos);
		// x(k+1) = x(k) + ��(k)p(k), r(k+1) = r(k) - ��(k)Ap(k)
		UpdateResidual << <dim3(ny, nz), nx >> > (residual, new_residual, p, Ap, f_pressure, alpha);
		// if ||r(k+1)|| is sufficient enough small, break
		printf("%d  %f  %f \n", i, aTb(new_residual, new_residual, max_pos));
		if (aTb(new_residual, new_residual, max_pos) < init_rTr * 1e-5)
			break;
		// ��(k) = r(k+1)Tr(k+1)/r(k)Tr(k)
		float beta = aTb(new_residual, new_residual, max_pos) / aTb(residual, residual, max_pos);
		// p(k+1) = r(k+1) + ��(k)p(k)
		UpdateP << <dim3(ny, nz), nx >> > (p, new_residual, beta);
		device_swap(&residual, &new_residual);
	}
}

void Solver::InitCuda()
{
	checkCudaErrors(hipMalloc((void**)&f_ux, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_uy, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_uz, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_new_ux, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_new_uy, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_new_uz, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_rho, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_new_rho, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_pressure, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_new_pressure, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_div, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_vortx, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_vorty, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_vortz, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&residual, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&new_residual, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&p, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&Ap, nx * ny * nz * sizeof(float)));

	checkCudaErrors(hipMemset(f_ux, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_uy, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_uz, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_new_ux, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_new_uy, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_new_uz, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_rho, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_new_rho, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_pressure, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_new_pressure, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_div, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_vortx, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_vorty, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_vortz, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(residual, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(new_residual, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(p, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(Ap, 0, nx * ny * nz * sizeof(float)));
}

void Solver::FreeCuda()
{
	checkCudaErrors(hipFree(f_ux));
	checkCudaErrors(hipFree(f_uy));
	checkCudaErrors(hipFree(f_uz));
	checkCudaErrors(hipFree(f_new_ux));
	checkCudaErrors(hipFree(f_new_uy));
	checkCudaErrors(hipFree(f_new_uz));
	checkCudaErrors(hipFree(f_rho));
	checkCudaErrors(hipFree(f_new_rho));
	checkCudaErrors(hipFree(f_pressure));
	checkCudaErrors(hipFree(f_new_pressure));
	checkCudaErrors(hipFree(f_div));
	checkCudaErrors(hipFree(f_vortx));
	checkCudaErrors(hipFree(f_vorty));
	checkCudaErrors(hipFree(f_vortz));
	checkCudaErrors(hipFree(residual));
	checkCudaErrors(hipFree(new_residual));
	checkCudaErrors(hipFree(p));
	checkCudaErrors(hipFree(Ap));
}

void Solver::UpdateCuda()
{
	float3 u;
	u.x = vel_x;
	u.y = vel_y;
	u.z = vel_z;

	int3 max_pos;
	max_pos.x = nx;
	max_pos.y = ny;
	max_pos.z = nz;

	// add source
	SourceKernel << <dim3(ny, nz), nx >> > (f_rho, f_ux, f_uy, f_uz, rho, u);
	// add force
	//VorticityKernel << <dim3(ny, nz), nx >> > (f_vortx, f_vorty, f_vortz, f_ux, f_uy, f_uz, max_pos);
	//ForceKernel << <dim3(ny, nz), nx >> > (f_ux, f_uy, f_uz, f_vortx, f_vorty, f_vortz, dt, curl_strength, max_pos);
	// velocity advection
	SemiLagKernel << <dim3(ny, nz), nx >> > (f_ux, f_new_ux, f_ux, f_uy, f_uz, dt, max_pos);
	SemiLagKernel << <dim3(ny, nz), nx >> > (f_uy, f_new_uy, f_ux, f_uy, f_uz, dt, max_pos);
	SemiLagKernel << <dim3(ny, nz), nx >> > (f_uz, f_new_uz, f_ux, f_uy, f_uz, dt, max_pos);
	swap(&f_ux, &f_new_ux);
	swap(&f_uy, &f_new_uy);
	swap(&f_uz, &f_new_uz);
	// density advection
	SemiLagKernel << <dim3(ny, nz), nx >> > (f_rho, f_new_rho, f_ux, f_uy, f_uz, dt, max_pos);
	swap(&f_rho, &f_new_rho);
	// divergence
	DivergenceKernel << <dim3(ny, nz), nx >> > (f_div, f_ux, f_uy, f_uz, max_pos);
#if 0
	// jacobi iteration
	for (int i = 0; i < max_iter; ++i)
	{
		JacobiKernel << <dim3(ny, nz), nx >> > (f_pressure, f_new_pressure, f_div, max_pos);
		swap(&f_pressure, &f_new_pressure);
	}
#else
	Conjugate << <dim3(1, 1), 1 >> > (residual, new_residual, p, Ap, f_pressure, f_div, max_pos);
#endif
	// update velocity
	ApplyGradient << <dim3(ny, nz), nx >> > (f_ux, f_uy, f_uz, f_pressure, max_pos);
}

void Solver::Initialize()
{
	f_density = (float*)malloc(nx * ny * nz * sizeof(float));
	InitCuda();
}

void Solver::Update()
{
	UpdateCuda();
}

Solver::~Solver()
{
	free(f_density);
	FreeCuda();
}

float* Solver::GetDensityField()
{
	checkCudaErrors(hipMemcpy(f_density, f_rho, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));
	return f_density;
}