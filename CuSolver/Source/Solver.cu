#include "hip/hip_runtime.h"
#define SOLVER_EXPORTS

#include "Solver.h"
#include "CudaUnitility.h"
#include <>

#define MGPCG 1

static __device__ int3 combine_int3(int a, int b, int c)
{
	int3 res;
	res.x = a, res.y = b, res.z = c;
	return res;
}

static __device__ float3 combine_float3(float a, float b, float c)
{
	float3 res;
	res.x = a, res.y = b, res.z = c;
	return res;
}

template<typename T>
static __device__ T min(T a, T b)
{
	return a < b ? a : b;
}

template<typename T>
static __device__ T max(T a, T b)
{
	return a > b ? a : b;
}

static __device__ int3 minmax(int3 pos, int3 max_pos)
{
	pos.x = max(0, min(pos.x, max_pos.x - 1));
	pos.y = max(0, min(pos.y, max_pos.y - 1));
	pos.z = max(0, min(pos.z, max_pos.z - 1));
	return pos;
}

static __device__ float sample(float* field, int3 pos, int3 max_pos)
{
	pos = minmax(pos, max_pos);
	return field[pos.x + pos.y * max_pos.x + pos.z * max_pos.x * max_pos.y];
}

static __device__ float neibor_sum(float* field, int pos, int offset, int3 max_pos)
{
	int i = pos % max_pos.x;
	int j = pos % (max_pos.x * max_pos.y) / max_pos.x;
	int k = pos / (max_pos.x * max_pos.y);
	return i > 0 ? field[offset + pos - 1] : 0 + i < max_pos.x - 1 ? field[offset + pos + 1] : 0 +
		j > 0 ? field[offset + pos - max_pos.x] : 0 + j < max_pos.y - 1 ? field[offset + pos + max_pos.x] : 0 +
		k > 0 ? field[offset + pos - max_pos.x * max_pos.y] : 0 + k < max_pos.z - 1 ? field[offset + pos + max_pos.x * max_pos.y] : 0;
}

static __device__ float cg_sample(float* field, int3 pos, int3 max_pos)
{
	if (pos.x < 0 || pos.x >= max_pos.x || pos.y < 0 || pos.y >= max_pos.y || pos.z < 0 || pos.z >= max_pos.z)
		return 0;
	return field[pos.x + pos.y * max_pos.x + pos.z * max_pos.x * max_pos.y];
}

static __device__ float length(float3 f)
{
	return sqrt(pow(f.x, 2) + pow(f.y, 2) + pow(f.z, 2));
}

static __device__ float3 normalize(float3 f)
{
	float len = length(f) + 1e-5;
	return combine_float3(f.x / len, f.y / len, f.z / len);
}

static __device__ float3 cross(float3 a, float3 b)
{
	return combine_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

static __device__ float lerp(float a, float b, float s)
{
	return a * (1 - s) + b * s;
}

static __device__ float trilerp(float* field, float3 pos, int3 max_pos)
{
	float x = pos.x;
	float y = pos.y;
	float z = pos.z;

	int ix = int(x);
	int iy = int(y);
	int iz = int(z);

	float fx = x - floor(x);
	float fy = y - floor(y);
	float fz = z - floor(z);

	float a = sample(field, combine_int3(ix, iy, iz), max_pos);
	float b = sample(field, combine_int3(ix + 1, iy, iz), max_pos);
	float c = sample(field, combine_int3(ix, iy + 1, iz), max_pos);
	float d = sample(field, combine_int3(ix + 1, iy + 1, iz), max_pos);
	float e = sample(field, combine_int3(ix, iy, iz + 1), max_pos);
	float f = sample(field, combine_int3(ix + 1, iy, iz + 1), max_pos);
	float g = sample(field, combine_int3(ix, iy + 1, iz + 1), max_pos);
	float h = sample(field, combine_int3(ix + 1, iy + 1, iz + 1), max_pos);

	float lerp1 = lerp(lerp(a, b, fx), lerp(c, d, fx), fy);
	float lerp2 = lerp(lerp(e, f, fx), lerp(g, h, fx), fy);

	return lerp(lerp1, lerp2, fz);
}

static __device__ float3 operator*(float a, float3 b)
{
	b.x *= a;
	b.y *= a;
	b.z *= a;
	return b;
}

static __device__ float3 operator*(float3 a, float3 b)
{
	b.x *= a.x;
	b.y *= a.y;
	b.z *= a.z;
	return b;
}

static __device__ float3 operator-(float3 a, float3 b)
{
	a.x -= b.x;
	a.y -= b.y;
	a.z -= b.z;
	return a;
}

static __device__ int3 operator*(int3 a, int b)
{
	a.x *= b;
	a.y *= b;
	a.z *= b;
	return a;
}

static __device__ int3 operator/(int3 a, int b)
{
	a.x /= b;
	a.y /= b;
	a.z /= b;
	return a;
}

static __device__ int3 operator+(int3 a, int3 b)
{
	a.x += b.x;
	a.y += b.y;
	a.z += b.z;
	return a;
}

static __device__ float3 RK2(float* ux, float* uy, float* uz, float3 pos, float dt, int3 max_pos)
{
	float3 u;
	u.x = trilerp(ux, pos, max_pos);
	u.y = trilerp(uy, pos, max_pos);
	u.z = trilerp(uz, pos, max_pos);
	float3 mid = pos - 0.5f * dt * u;
	u.x = trilerp(ux, mid, max_pos);
	u.y = trilerp(uy, mid, max_pos);
	u.z = trilerp(uz, mid, max_pos);
	// here may exist out of range problem
	return pos - dt * u;
}

void swap(float** a, float** b)
{
	float* temp = *a;
	*a = *b;
	*b = temp;
}

static __global__ void CopyFrom(float* dst, float* src)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	dst[ind] = src[ind];
}

static __global__ void Fill(float* field, int offset, float fill)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	field[offset + ind] = fill;
}

static __global__ void GlobalReduce(float* a, float* b, float* res)
{
	 __shared__ float sdata[1024];
	size_t tid = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = tid + j * blockDim.x + k * blockDim.x * gridDim.x;

	sdata[tid] = a[ind] * b[ind];
	__syncthreads();

	// do reduction in shared memory
	for (unsigned int s = blockDim.x / 2; s > 32; s >>= 1)
	{
		if (tid < s)
			sdata[tid] += sdata[tid + s];
		__syncthreads();
	}
	// when threads <=32, there is only one wrap is working, no synchonization is required in a wrap
	// there are still some optimization, maybe applied later
	if (tid < 32)
	{
		sdata[tid] += sdata[tid + 32]; sdata[tid] += sdata[tid + 16];
		sdata[tid] += sdata[tid + 8]; sdata[tid] += sdata[tid + 4];
		sdata[tid] += sdata[tid + 2]; sdata[tid] += sdata[tid + 1];
	}

	// write result to global memory
	if (tid == 0)
		res[blockIdx.x + blockIdx.y * gridDim.x] = sdata[0];
}

static __global__ void BlockReduce(float* a)
{
	size_t ind = threadIdx.x;

	for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1)
	{
		if (ind < s)
			a[ind] += a[ind + s];
		__syncthreads();
	}
}

static __global__ void SourceKernel(float* rho, float* ux, float* uy, float* uz, float rho0, float3 u0)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	if (i > blockDim.x / 2 - 2 && i < blockDim.x / 2 + 2 && j > gridDim.x / 2 - 2 && j < gridDim.x / 2 + 2 && k > 1 && k < 3)
	{
		rho[ind] = rho0;
		ux[ind] = u0.x;
		uy[ind] = u0.y;
		uz[ind] = u0.z;
	}
}

static __global__ void SemiLagKernel(float* field, float* new_field, float* ux, float* uy, float* uz, float dt, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float3 pos;
	pos.x = float(i);
	pos.y = float(j);
	pos.z = float(k);

	float3 coord = RK2(ux, uy, uz, pos, dt, max_pos);
	new_field[ind] = trilerp(field, coord, max_pos);
}

static __global__ void DivergenceKernel(float* field, float* ux, float* uy, float* uz, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float ul = sample(ux, combine_int3(i - 1, j, k), max_pos);
	float ur = sample(ux, combine_int3(i + 1, j, k), max_pos);
	float ubh = sample(uy, combine_int3(i, j - 1, k), max_pos);
	float uf = sample(uy, combine_int3(i, j + 1, k), max_pos);
	float ubo = sample(uz, combine_int3(i, j, k - 1), max_pos);
	float ut = sample(uz, combine_int3(i, j, k + 1), max_pos);

#if 0
	// box boundary
	float ucx = sample(ux, combine_int3(i, j, k), max_pos);
	float ucy = sample(uy, combine_int3(i, j, k), max_pos);
	float ucz = sample(uz, combine_int3(i, j, k), max_pos);
	if (i == 0)
		ul = -ucx;
	if (i == max_pos.x - 1)
		ur = -ucx;
	if (j == 0)
		ubh = -ucy;
	if (j == max_pos.y - 1)
		uf = -ucy;
	if (k == 0)
		ubo = -ucz;
	if (k == max_pos.z - 1)
		ut = -ucz;
#endif

	float div = (ur + uf + ut - ul - ubh - ubo) * 0.5;

	field[ind] = div;
}

static __global__ void JacobiKernel(float* field, float* new_field, float* div_field, float* r, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float pl = sample(field, combine_int3(i - 1, j, k), max_pos);
	float pr = sample(field, combine_int3(i + 1, j, k), max_pos);
	float pbh = sample(field, combine_int3(i, j - 1, k), max_pos);
	float pf = sample(field, combine_int3(i, j + 1, k), max_pos);
	float pbo = sample(field, combine_int3(i, j, k - 1), max_pos);
	float pt = sample(field, combine_int3(i, j, k + 1), max_pos);
	float div = sample(div_field, combine_int3(i, j, k), max_pos);

	new_field[ind] = (pl + pr + pbh + pf + pbo + pt - div) / 6.f;
	r[ind] = div + 6 * field[ind] - pl - pr - pbh - pf - pbo - pt;
}

static __global__ void ApplyGradient(float* f_ux, float* f_uy, float* f_uz, float* pressure_field, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float pl = sample(pressure_field, combine_int3(i - 1, j, k), max_pos);
	float pr = sample(pressure_field, combine_int3(i + 1, j, k), max_pos);
	float pbh = sample(pressure_field, combine_int3(i, j - 1, k), max_pos);
	float pf = sample(pressure_field, combine_int3(i, j + 1, k), max_pos);
	float pbo = sample(pressure_field, combine_int3(i, j, k - 1), max_pos);
	float pt = sample(pressure_field, combine_int3(i, j, k + 1), max_pos);

	f_ux[ind] -= 0.5 * (pr - pl);
	f_uy[ind] -= 0.5 * (pf - pbh);
	f_uz[ind] -= 0.5 * (pt - pbo);
}

static __global__ void VorticityKernel(float* f_vortx, float* f_vorty, float* f_vortz, float* ux, float* uy, float* uz, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float ul = sample(ux, combine_int3(i - 1, j, k), max_pos);
	float ur = sample(ux, combine_int3(i + 1, j, k), max_pos);
	float ubh = sample(uy, combine_int3(i, j - 1, k), max_pos);
	float uf = sample(uy, combine_int3(i, j + 1, k), max_pos);
	float ubo = sample(uz, combine_int3(i, j, k - 1), max_pos);
	float ut = sample(uz, combine_int3(i, j, k + 1), max_pos);

	f_vortx[ind] = (uf - ubh - ut + ubo) * 0.5;
	f_vorty[ind] = (ut - ubo - ur + ul) * 0.5;
	f_vortz[ind] = (ur - ul - uf + ubh) * 0.5;
}

static __global__ void ForceKernel(float* f_ux, float* f_uy, float* f_uz, float* f_vortx, float* f_vorty, float* f_vortz, float dt, float curl_strength, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float3 vl = combine_float3(sample(f_vortx, combine_int3(i - 1, j, k), max_pos),
		sample(f_vorty, combine_int3(i - 1, j, k), max_pos),
		sample(f_vortz, combine_int3(i - 1, j, k), max_pos));
	float3 vr = combine_float3(sample(f_vortx, combine_int3(i + 1, j, k), max_pos),
		sample(f_vorty, combine_int3(i + 1, j, k), max_pos),
		sample(f_vortz, combine_int3(i + 1, j, k), max_pos));
	float3 vbh = combine_float3(sample(f_vortx, combine_int3(i, j - 1, k), max_pos),
		sample(f_vorty, combine_int3(i, j - 1, k), max_pos),
		sample(f_vortz, combine_int3(i, j - 1, k), max_pos));
	float3 vf = combine_float3(sample(f_vortx, combine_int3(i, j + 1, k), max_pos),
		sample(f_vorty, combine_int3(i, j + 1, k), max_pos),
		sample(f_vortz, combine_int3(i, j + 1, k), max_pos));
	float3 vbo = combine_float3(sample(f_vortx, combine_int3(i, j, k - 1), max_pos),
		sample(f_vorty, combine_int3(i, j, k - 1), max_pos),
		sample(f_vortz, combine_int3(i, j, k - 1), max_pos));
	float3 vt = combine_float3(sample(f_vortx, combine_int3(i, j, k + 1), max_pos),
		sample(f_vorty, combine_int3(i, j, k + 1), max_pos),
		sample(f_vortz, combine_int3(i, j, k + 1), max_pos));
	float3 vc = combine_float3(sample(f_vortx, combine_int3(i, j, k), max_pos),
		sample(f_vorty, combine_int3(i, j, k), max_pos),
		sample(f_vortz, combine_int3(i, j, k), max_pos));

	// �� = ��|��|, N = ��/|��|
	float3 force = normalize(combine_float3(abs(length(vr)) - abs(length(vl)), abs(length(vf)) - abs(length(vbh)), abs(length(vt)) - abs(length(vbo))));
	// f_conf(vort) = ��h(N����)
	float3 fvort = curl_strength * cross(force, vc);

	f_ux[ind] += fvort.x * dt;
	f_uy[ind] += fvort.y * dt;
	f_uz[ind] += fvort.z * dt;
}

// -Ax = -b, r0 = -b = -����u
static __global__ void InitConjugate(float* r, float* f_div, float* x)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	r[ind] = -f_div[ind];
	x[ind] = 0;
}

// p here is conjugate gradient, not pressure
static __global__ void ComputeAp(float* Ap, float* p, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float pc = cg_sample(p, combine_int3(i, j, k), max_pos);

	Ap[ind] = 6.f * pc - neibor_sum(p, ind, 0, max_pos);
	//printf("ap %f %f\n", pc, neibor_sum(p, ind, 0, max_pos));
}

static __global__ void UpdateResidual(float* r, float* p, float* Ap, float* x, float alpha)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	x[ind] += alpha * p[ind];
	r[ind] -= alpha * Ap[ind];
}

static __global__ void UpdateP(float* p, float* z, float beta)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	p[ind] = z[ind] + beta * p[ind];
}

static __global__ void Restrict(float* r, float* z, int offset, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	float res = r[offset + ind] - (6 * z[offset + ind] - neibor_sum(z, ind, offset, max_pos));
	// r[l+1][pos//2] += res * 0.5
	offset += max_pos.x * max_pos.y * max_pos.z;
	int new_ind = i >> 2 + (j >> 2) * (max_pos.x >> 2) + (k >> 2) * (max_pos.x >> 2) * (max_pos.y >> 2);
	r[offset + new_ind] += res * 0.5;
}

static __global__ void Prolongate(float* z, int offset, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	// r[l][pos] = r[l+1][pos//2]
	int new_ind = i >> 2 + (j >> 2) * (max_pos.x >> 2) + (k >> 2) * (max_pos.x >> 2) * (max_pos.y >> 2);
	z[offset + ind] += z[offset + max_pos.x * max_pos.y * max_pos.z + new_ind];
}

static __global__ void Smooth(float* r, float* z, int offset, bool phase, int3 max_pos)
{
	size_t i = threadIdx.x;
	size_t j = blockIdx.x;
	size_t k = blockIdx.y;
	size_t ind = i + j * blockDim.x + k * blockDim.x * gridDim.x;

	// red/black Gauss Seidel
	if (bool((i + j + k) & 1) == phase)
	{
		z[offset + ind] = (r[offset + ind] + neibor_sum(z, ind, offset, max_pos)) / 6.f;
	}
}

void Solver::InitCuda()
{
	checkCudaErrors(hipMalloc((void**)&f_ux, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_uy, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_uz, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_new_ux, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_new_uy, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_new_uz, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_rho, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_new_rho, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_pressure, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_new_pressure, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_div, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_vortx, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_vorty, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&f_vortz, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&r, mg_space * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&z, mg_space * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&new_z, mg_space * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&p, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&Ap, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&x, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&temp, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMalloc((void**)&d_temp_res, sizeof(float)));

	checkCudaErrors(hipMemset(f_ux, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_uy, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_uz, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_new_ux, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_new_uy, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_new_uz, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_rho, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_new_rho, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_pressure, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_new_pressure, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_div, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_vortx, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_vorty, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(f_vortz, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(r, 0, mg_space * sizeof(float)));
	checkCudaErrors(hipMemset(z, 0, mg_space * sizeof(float)));
	checkCudaErrors(hipMemset(new_z, 0, mg_space * sizeof(float)));
	checkCudaErrors(hipMemset(p, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(Ap, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(x, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(temp, 0, nx * ny * nz * sizeof(float)));
	checkCudaErrors(hipMemset(d_temp_res, 0, sizeof(float)));
}

void Solver::FreeCuda()
{
	checkCudaErrors(hipFree(f_ux));
	checkCudaErrors(hipFree(f_uy));
	checkCudaErrors(hipFree(f_uz));
	checkCudaErrors(hipFree(f_new_ux));
	checkCudaErrors(hipFree(f_new_uy));
	checkCudaErrors(hipFree(f_new_uz));
	checkCudaErrors(hipFree(f_rho));
	checkCudaErrors(hipFree(f_new_rho));
	checkCudaErrors(hipFree(f_pressure));
	checkCudaErrors(hipFree(f_new_pressure));
	checkCudaErrors(hipFree(f_div));
	checkCudaErrors(hipFree(f_vortx));
	checkCudaErrors(hipFree(f_vorty));
	checkCudaErrors(hipFree(f_vortz));
	checkCudaErrors(hipFree(r));
	checkCudaErrors(hipFree(z));
	checkCudaErrors(hipFree(new_z));
	checkCudaErrors(hipFree(p));
	checkCudaErrors(hipFree(Ap));
	checkCudaErrors(hipFree(x));
	checkCudaErrors(hipFree(temp));
	checkCudaErrors(hipFree(d_temp_res));
}

void Solver::UpdateCuda()
{
	float3 u;
	u.x = vel_x;
	u.y = vel_y;
	u.z = vel_z;

	int3 max_pos;
	max_pos.x = nx;
	max_pos.y = ny;
	max_pos.z = nz;

	// add source
	SourceKernel << <dim3(ny, nz), nx >> > (f_rho, f_ux, f_uy, f_uz, rho, u);
	// add force
	VorticityKernel << <dim3(ny, nz), nx >> > (f_vortx, f_vorty, f_vortz, f_ux, f_uy, f_uz, max_pos);
	ForceKernel << <dim3(ny, nz), nx >> > (f_ux, f_uy, f_uz, f_vortx, f_vorty, f_vortz, dt, curl_strength, max_pos);
	// velocity advection
	SemiLagKernel << <dim3(ny, nz), nx >> > (f_ux, f_new_ux, f_ux, f_uy, f_uz, dt, max_pos);
	SemiLagKernel << <dim3(ny, nz), nx >> > (f_uy, f_new_uy, f_ux, f_uy, f_uz, dt, max_pos);
	SemiLagKernel << <dim3(ny, nz), nx >> > (f_uz, f_new_uz, f_ux, f_uy, f_uz, dt, max_pos);
	swap(&f_ux, &f_new_ux);
	swap(&f_uy, &f_new_uy);
	swap(&f_uz, &f_new_uz);
	// density advection
	SemiLagKernel << <dim3(ny, nz), nx >> > (f_rho, f_new_rho, f_ux, f_uy, f_uz, dt, max_pos);
	swap(&f_rho, &f_new_rho);
	// divergence
	DivergenceKernel << <dim3(ny, nz), nx >> > (f_div, f_ux, f_uy, f_uz, max_pos);
#if 0
	// jacobi iteration
	for (int i = 0; i < max_iter; ++i)
	{
		JacobiKernel << <dim3(ny, nz), nx >> > (f_pressure, f_new_pressure, f_div, r, max_pos);
		swap(&f_pressure, &f_new_pressure);
		//aTb << <1, 1 >> > (r, r,d_temp_res, max_pos);
		//checkCudaErrors(hipMemcpy(&rTr, d_temp_res, sizeof(float), hipMemcpyDeviceToHost));
		//std::cout << "iter " << i << " rTr: " << rTr << std::endl;
	}
#else
	Conjugate();
#endif
	// update velocity
	ApplyGradient << <dim3(ny, nz), nx >> > (f_ux, f_uy, f_uz, f_pressure, max_pos);
}

void Solver::Initialize()
{
	f_density = (float*)malloc(nx * ny * nz * sizeof(float));

	// note that the layout of our multi level grid is [level][z][y][x]
	int temp_space = nx * ny * nz;
	for (int i = 0; i < mg_level; ++i)
	{
		mg_space += temp_space;
		temp_space /= 8;
	}

	InitCuda();
}

void Solver::Update()
{
	printf("frame: %d\n", current_frame);
	UpdateCuda();
	current_frame++;
}

Solver::~Solver()
{
	free(f_density);
	FreeCuda();
}

void Solver::Conjugate()
{
	int3 max_pos;
	max_pos.x = nx;
	max_pos.y = ny;
	max_pos.z = nz;

	int n = nx * ny * nz;

	InitConjugate << <dim3(ny, nz), nx >> > (r, f_div, x);

	// aTb operator, calc the sum of each block and then reduce all the data
	// note that the number of thread in each block cannot exceed 1024
	// the number here should self-adapte to the amount of euler girds, here is 64*64*256, should be modified whenever grid size is changed
	GlobalReduce << <dim3(32, 32), 1024 >> > (r, r, temp);
	BlockReduce << <1, 1024 >> > (temp);
	checkCudaErrors(hipMemcpy(&init_rTr, &temp[0], sizeof(float), hipMemcpyDeviceToHost));

	std::cout << "init rTr: " << init_rTr << std::endl;

#if MGPCG
	MG_Preconditioner();
#else
	CopyFrom << <dim3(ny, nz), nx >> > (z, r);
#endif

	// p(0) = M^-1 r(0)
	UpdateP << <dim3(ny, nz), nx >> > (p, z, 0);

	GlobalReduce << <dim3(32, 32), 1024 >> > (z, r, temp);
	BlockReduce << <1, 1024 >> > (temp);
	checkCudaErrors(hipMemcpy(&old_zTr, &temp[0], sizeof(float), hipMemcpyDeviceToHost));

	for (int i = 0; i < 30; ++i)
	{
		// ��(k) = r(k)Tr(k) / p(k)TAp(k)
		ComputeAp << <dim3(ny, nz), nx >> > (Ap, p, max_pos);
		GlobalReduce << <dim3(32, 32), 1024 >> > (p, Ap, temp);
		BlockReduce << <1, 1024 >> > (temp);
		checkCudaErrors(hipMemcpy(&pAp, &temp[0], sizeof(float), hipMemcpyDeviceToHost));
		float alpha = old_zTr / pAp;

		// x(k+1) = x(k) + ��(k)p(k), r(k+1) = r(k) - ��(k)Ap(k)
		UpdateResidual << <dim3(ny, nz), nx >> > (r, p, Ap, x, alpha);

		// if ||r(k+1)|| is sufficient enough small, break
		GlobalReduce << <dim3(32, 32), 1024 >> > (r, r, temp);
		BlockReduce << <1, 1024 >> > (temp);
		checkCudaErrors(hipMemcpy(&rTr, &temp[0], sizeof(float), hipMemcpyDeviceToHost));
		std::cout << "iter " << i << " rTr: " << rTr << std::endl;

		// early stop
		if (rTr < init_rTr * 1e-14 || rTr * 10 > last_rTr || rTr == 0)
			break;

#if MGPCG
		MG_Preconditioner();
#else
		CopyFrom << <dim3(ny, nz), nx >> > (z, r);
#endif

		// ��(k) = r(k+1)Tr(k+1)/r(k)Tr(k)
		GlobalReduce << <dim3(32, 32), 1024 >> > (z, r, temp);
		BlockReduce << <1, 1024 >> > (temp);
		checkCudaErrors(hipMemcpy(&new_zTr, &temp[0], sizeof(float), hipMemcpyDeviceToHost));
		float beta = new_zTr / old_zTr;
		// p(k+1) = r(k+1) + ��(k)p(k)
		UpdateP << <dim3(ny, nz), nx >> > (p, z, beta);

		old_zTr = new_zTr;
		last_rTr = rTr;
	}

	CopyFrom << <dim3(ny, nz), nx >> > (f_pressure, x);
}

void Solver::MG_Preconditioner()
{
	int3 max_pos;
	max_pos.x = nx;
	max_pos.y = ny;
	max_pos.z = nz;

	int r_offset = nx * ny * nz;
	int offset = 0;

	// initialize z[l] and r[l] with 0 except r[0]
	Fill << <1, mg_space >> > (z, 0, 0);
	Fill << <1, mg_space - r_offset >> > (r, r_offset, 0);	

	// downsample
	for (int l = 0; l < mg_level - 1; ++l)
	{
		for (int i = 0; i < init_smooth_steps << l; ++i)
		{
			Smooth << <dim3(max_pos.y, max_pos.z), max_pos.x >> > (r, z, offset, 0, max_pos);
			Smooth << <dim3(max_pos.y, max_pos.z), max_pos.x >> > (r, z, offset, 1, max_pos);
		}
		Restrict << <dim3(max_pos.y, max_pos.z), max_pos.x >> > (r, z, offset, max_pos);

		offset += max_pos.x * max_pos.y * max_pos.z;
		max_pos.x = max_pos.x >> 1;
		max_pos.y = max_pos.y >> 1;
		max_pos.z = max_pos.z >> 1;
	}

	// bottom smoothing
	for (int i = 0; i < bottom_smooth_steps; ++i)
	{
		Smooth << <dim3(max_pos.y, max_pos.z), max_pos.x >> > (r, z, offset, 0, max_pos);
		Smooth << <dim3(max_pos.y, max_pos.z), max_pos.x >> > (r, z, offset, 1, max_pos);
	}
	// upsample
	for (int l = mg_level - 2; l >= 0; --l)
	{
		max_pos.x = max_pos.x << 1;
		max_pos.y = max_pos.y << 1;
		max_pos.z = max_pos.z << 1;
		offset -= max_pos.x * max_pos.y * max_pos.z;
		Prolongate << <dim3(max_pos.y, max_pos.z), max_pos.x >> > (z, offset, max_pos);

		for (int i = 0; i < init_smooth_steps << l; ++i)
		{
			Smooth << <dim3(max_pos.y, max_pos.z), max_pos.x >> > (r, z, offset, 0, max_pos);
			Smooth << <dim3(max_pos.y, max_pos.z), max_pos.x >> > (r, z, offset, 1, max_pos);
		}
	}
}

float* Solver::GetDensityField()
{
	checkCudaErrors(hipMemcpy(f_density, f_rho, nx * ny * nz * sizeof(float), hipMemcpyDeviceToHost));
	return f_density;
}